#include "hip/hip_runtime.h"
﻿#include "pathtrace.h"

#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <iostream>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/sort.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/tuple.h>
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include <stream_compaction/efficient.cu>
#include <OpenImageDenoise/oidn.hpp>

#include "sceneStructs.h"
#include "scene.h"
#include "utilities.h"
#include "intersections.h"
#include "interactions.h"

#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)

#define STREAMCOMPACTION 1
#define MATERIALSORTING 1
#define BVH 1;
#define ACES 1;
#define REINHARD 0;
#define GAMMACORRECTION 1;
#define RUSSIANROULETTE 1;
#define OIDN 1;

void checkCUDAErrorFn(const char* msg, const char* file, int line)
{
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err)
    {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file)
    {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#ifdef _WIN32
    getchar();
#endif // _WIN32
    exit(EXIT_FAILURE);
#endif // ERRORCHECK
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth)
{
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution, int iter, glm::vec3* image)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y)
    {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        glm::vec3 postReinhard;

        pix.x /= iter;
        pix.y /= iter;
        pix.z /= iter;

#if REINHARD
        // Reinhard Operator
        pix.x = pix.x / (pix.x + 1);
        pix.y = pix.y / (pix.y + 1);
        pix.z = pix.z / (pix.z + 1);
#endif

#if ACES
        float a = 2.51f;
        float b = 0.03f;
        float c = 2.43f;
        float d = 0.59f;
        float e = 0.14f;
        pix = (pix * (a * pix + b)) / (pix * (c * pix + d) + e);
#endif

#if GAMMACORRECTION
        pix.x = glm::pow(pix.x, 1.0 / 2.2);
        pix.y = glm::pow(pix.y, 1.0 / 2.2);
        pix.z = glm::pow(pix.z, 1.0 / 2.2);
#endif 
        color.x = glm::clamp((int)(pix.x * 255.0), 0, 255);
        color.y = glm::clamp((int)(pix.y * 255.0), 0, 255);
        color.z = glm::clamp((int)(pix.z * 255.0), 0, 255);;

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

static Scene* hst_scene = NULL;
static GuiDataContainer* guiData = NULL;
static glm::vec3* dev_image = NULL;
static glm::vec3* dev_image_albedo = NULL;
static glm::vec3* dev_image_normal = NULL;
static glm::vec3* dev_image_denoised = NULL;
static Geom* dev_geoms = NULL;
static Material* dev_materials = NULL;
static PathSegment* dev_paths = NULL;
static ShadeableIntersection* dev_intersections = NULL;
// TODO: static variables for device memory, any extra info you need, etc
// ...
static int* dev_isValidIntersection = NULL;
static DeviceTexture* dev_textures = NULL;

void InitDataContainer(GuiDataContainer* imGuiData)
{
    guiData = imGuiData;
}

void pathtraceInit(Scene* scene)
{
    hst_scene = scene;

    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

    // Checkpointed image copy
    printf("Init called with iter: %ui", hst_scene->state.currIteration);
    if (hst_scene->state.currIteration != 0) {
        hipMemcpy(dev_image, hst_scene->state.image.data(),
            pixelcount * sizeof(glm::vec3), hipMemcpyHostToDevice);
    }

    hipMalloc(&dev_image_albedo, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image_albedo, 0, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_image_normal, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image_normal, 0, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_image_denoised, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image_denoised, 0, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

    printf("INITING");

    // Copy all mesh data onto the GPU
    checkCUDAError("other");
    for (Geom& geom : scene->geoms) {
        if (geom.type == MESH && !geom.mesh.onGpu) {
            Mesh& mesh = geom.mesh;

            checkCUDAError("other");
            glm::vec3 *posTmp, *norTmp;
            glm::vec2* uvTmp;
            unsigned short* indTmp, *indBvhTmp;
            BvhNode* nodesTmp;

            checkCUDAError("other");
            hipMalloc((void**)&posTmp, mesh.posCount * sizeof(glm::vec3));
            hipMemcpy(posTmp, mesh.pos, mesh.posCount * sizeof(glm::vec3), hipMemcpyHostToDevice);
            free(mesh.pos);
            mesh.pos = posTmp;

            checkCUDAError("other");
            hipMalloc((void**)&norTmp, mesh.norCount * sizeof(glm::vec3));
            hipMemcpy(norTmp, mesh.nor, mesh.norCount * sizeof(glm::vec3), hipMemcpyHostToDevice);
            free(mesh.nor);
            mesh.nor = norTmp;


            size_t size = mesh.uvCount * sizeof(float3);
            std::cout << "Allocating " << size << " bytes (" << (size / (1024.0 * 1024.0)) << " MB)\n";
            checkCUDAError("other");
            hipMalloc((void**)&uvTmp, mesh.uvCount * sizeof(glm::vec2));
            hipMemcpy(uvTmp, mesh.uv, mesh.uvCount * sizeof(glm::vec2), hipMemcpyHostToDevice);
            free(mesh.uv);
            mesh.uv = uvTmp;

            checkCUDAError("other");
            hipMalloc((void**)&indTmp, mesh.indCount * sizeof(unsigned short));
            hipMemcpy(indTmp, mesh.ind, mesh.indCount * sizeof(unsigned short), hipMemcpyHostToDevice);
            free(mesh.ind);
            mesh.ind = indTmp;

            if (mesh.numBvhNodes > 0) {
                hipMalloc((void**)&nodesTmp, mesh.numBvhNodes * sizeof(BvhNode));
                hipMemcpy(nodesTmp, mesh.bvhNodes, mesh.numBvhNodes * sizeof(BvhNode), hipMemcpyHostToDevice);
                free(mesh.bvhNodes);
                mesh.bvhNodes = nodesTmp;

                hipMalloc((void**)&indBvhTmp, mesh.indCount * sizeof(unsigned short));
                hipMemcpy(indBvhTmp, mesh.indBVH, mesh.indCount * sizeof(unsigned short), hipMemcpyHostToDevice);
                free(mesh.indBVH);
                mesh.indBVH = indBvhTmp;
            }

            geom.mesh.onGpu = true;
            printf("LOADED ONTO THE GPU");
            hipDeviceSynchronize();
        }
    }

    hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
    hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

    hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
    hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

    hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
    hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    // TODO: initialize any extra device memeory you need
    checkCUDAError("other");
    hipMalloc(&dev_isValidIntersection, pixelcount * sizeof(int));
    hipMemset(dev_isValidIntersection, 0, pixelcount * sizeof(int));

    // We will assemble a new textures array here to put on the GPU 
    // because I believe std::vectors cannot go onto the GPU.
    std::vector<DeviceTexture> dev_textureArr;

    // Put all textures onto the GPU
    // TODO: Will this work with 3 channel textures or will I need to pad?
    for (Texture& tex : scene->textures) {
        if (tex.numChannels < 1 || tex.numChannels > 4) {
            throw std::runtime_error("Unsupported channel count");
        }
        if (tex.bitsPerChannel != 8 && tex.bitsPerChannel != 16 && tex.bitsPerChannel != 32) {
            throw std::runtime_error("Unsupported bit depth (only 8, 16 or 32)");
        }

        // 1. Create channel descriptor
        checkCUDAError("other");
        hipChannelFormatDesc channelDesc;
        if (tex.bitsPerChannel == 8) {
            channelDesc = hipCreateChannelDesc(
                (tex.numChannels >= 1) ? 8 : 0,
                (tex.numChannels >= 2) ? 8 : 0,
                (tex.numChannels >= 3) ? 8 : 0,
                (tex.numChannels >= 4) ? 8 : 0,
                hipChannelFormatKindUnsigned);
        }
        else if (tex.bitsPerChannel == 16) {
            channelDesc = hipCreateChannelDesc(
                (tex.numChannels >= 1) ? 16 : 0,
                (tex.numChannels >= 2) ? 16 : 0,
                (tex.numChannels >= 3) ? 16 : 0,
                (tex.numChannels >= 4) ? 16 : 0,
                hipChannelFormatKindUnsigned);
        }
        else { // 32-bit float
            channelDesc = hipCreateChannelDesc(
                (tex.numChannels >= 1) ? 32 : 0,
                (tex.numChannels >= 2) ? 32 : 0,
                (tex.numChannels >= 3) ? 32 : 0,
                (tex.numChannels >= 4) ? 32 : 0,
                hipChannelFormatKindFloat);
        }
        checkCUDAError("Create Channel Desc");

        // 2. Allocate CUDA array
        hipArray_t cuArray;
        hipMallocArray(&cuArray, &channelDesc, tex.width, tex.height);

        // 3. Copy host → device array
        size_t elemSize = tex.bitsPerChannel / 8;
        size_t rowSizeBytes = tex.width * tex.numChannels * elemSize;
        checkCUDAError("Create Channel Desc2");

        hipMemcpy2DToArray(
            cuArray, 0, 0,
            tex.data.data(), rowSizeBytes,
            rowSizeBytes, tex.height,
            hipMemcpyHostToDevice);
        checkCUDAError("Copy texture to GPU");

        // 4. Resource description
        hipResourceDesc resDesc;
        memset(&resDesc, 0, sizeof(resDesc));
        resDesc.resType = hipResourceTypeArray;
        resDesc.res.array.array = cuArray;

        // 5. Texture description
        hipTextureDesc texDesc;
        memset(&texDesc, 0, sizeof(texDesc));
        texDesc.addressMode[0] = hipAddressModeClamp;
        texDesc.addressMode[1] = hipAddressModeClamp;
        texDesc.filterMode = hipFilterModePoint;
        texDesc.readMode = hipReadModeNormalizedFloat;
        if (tex.bitsPerChannel == 32) {
            texDesc.readMode = hipReadModeElementType; // If float type, do not normalize
        }
        texDesc.normalizedCoords = 1;             

        // 6. Create texture object
        hipTextureObject_t texObj = 0;
        hipCreateTextureObject(&texObj, &resDesc, &texDesc, nullptr);
        checkCUDAError("Create texture handle");

        tex.d_data = cuArray;
        tex.d_texHandle = texObj;

        DeviceTexture dTex;
        dTex.texHandle = texObj;
        dTex.width = tex.width;
        dTex.height = tex.height;
        dev_textureArr.push_back(dTex);
    }
    
    hipMalloc(&dev_textures, dev_textureArr.size()* sizeof(DeviceTexture));
    hipMemcpy(dev_textures, dev_textureArr.data(), dev_textureArr.size() * sizeof(DeviceTexture), hipMemcpyHostToDevice);

    checkCUDAError("pathtraceInit");
}

void pathtraceFree(Scene* scene)
{
    hipFree(dev_image);  // no-op if dev_image is null
    hipFree(dev_image_albedo);
    hipFree(dev_image_normal);
    hipFree(dev_image_denoised);
    hipFree(dev_paths);
    checkCUDAError("pathtraceFree1");

    // Free individual mesh buffers
    for (Geom& geom : scene->geoms) {
        if (geom.type == MESH && geom.mesh.onGpu) {
            Mesh& mesh = geom.mesh;

            glm::vec3* posTmp = new glm::vec3[mesh.posCount];
            glm::vec3* norTmp = new glm::vec3[mesh.norCount];
            glm::vec2* uvTmp = new glm::vec2[mesh.uvCount];
            unsigned short* indTmp = new unsigned short[mesh.indCount];
            BvhNode* nodesTmp = new BvhNode[mesh.numBvhNodes];
            unsigned short* indBvhTmp = new unsigned short[mesh.indCount];

            hipMemcpy(posTmp, mesh.pos, mesh.posCount * sizeof(glm::vec3), hipMemcpyDeviceToHost);
            hipFree(mesh.pos);
            checkCUDAError("pathtraceFree2");
            mesh.pos = posTmp;

            hipMemcpy(norTmp, mesh.nor, mesh.norCount * sizeof(glm::vec3), hipMemcpyDeviceToHost);
            hipFree(mesh.nor);
            checkCUDAError("pathtraceFree2");
            mesh.nor = norTmp;

            hipMemcpy(uvTmp, mesh.uv, mesh.uvCount * sizeof(glm::vec2), hipMemcpyDeviceToHost);
            hipFree(mesh.uv);
            checkCUDAError("pathtraceFree2");
            mesh.uv = uvTmp;

            hipMemcpy(indTmp, mesh.ind, mesh.indCount * sizeof(unsigned short), hipMemcpyDeviceToHost);
            hipFree(mesh.ind);
            checkCUDAError("pathtraceFree2");
            mesh.ind = indTmp;

            if (mesh.numBvhNodes > 0) {
                hipMemcpy(nodesTmp, mesh.bvhNodes, mesh.numBvhNodes * sizeof(BvhNode), hipMemcpyDeviceToHost);
                hipFree(mesh.bvhNodes);
                checkCUDAError("pathtraceFree2");
                mesh.bvhNodes = nodesTmp;

                hipMemcpy(indBvhTmp, mesh.indBVH, mesh.indCount * sizeof(unsigned short), hipMemcpyDeviceToHost);
                hipFree(mesh.indBVH);
                checkCUDAError("pathtraceFree2");
                mesh.indBVH = indBvhTmp;
            }

            geom.mesh.onGpu = false;
            printf("LOADED BACK ONTO CPU");
            hipDeviceSynchronize();

        }
    }
    checkCUDAError("pathtraceFree2");

    // Free textures
    for (Texture& tex : scene->textures) {
        if (tex.d_texHandle) {
            hipDestroyTextureObject(tex.d_texHandle);
            tex.d_texHandle = 0;
        }
        if (tex.d_data) {
            hipFreeArray(tex.d_data);
            tex.d_data = nullptr;
        }
    }

    hipFree(dev_geoms);
    hipFree(dev_materials);
    hipFree(dev_intersections);
    // TODO: clean up any extra device memory you created
    hipFree(dev_isValidIntersection);
    hipFree(dev_textures);

    checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < cam.resolution.x && y < cam.resolution.y) {
        int index = x + (y * cam.resolution.x);
        PathSegment& segment = pathSegments[index];

        segment.ray.origin = cam.position;
        segment.radiance = glm::vec3(0.0f, 0.0f, 0.0f);
        segment.throughput = glm::vec3(1.0f, 1.0f, 1.0f);
        segment.firstAlbedo = glm::vec3(-1.f);
        segment.firstNormal = glm::vec3(-1.f);


        thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, traceDepth);
        thrust::uniform_real_distribution<float> u01(0, 1);
        segment.ray.direction = glm::normalize(cam.view
            - cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f)
            - cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f)
            + cam.right * cam.pixelLength.x * (u01(rng) - 0.5f)
            + cam.up * cam.pixelLength.y * (u01(rng) - 0.5f)
        );

        // DOF implementation
        //TODO: attach to ui
        float focal_distance = 2.f;
        glm::vec3 target = cam.view
            - cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f)
            - cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f)
            + cam.right * cam.pixelLength.x * (u01(rng) - 0.5f)
            + cam.up * cam.pixelLength.y * (u01(rng) - 0.5f);
        target *= focal_distance;

        //TODO: attach to ui
        float dofScale = 0.0f;
        glm::vec3 posOffset = dofScale * cam.right * (u01(rng) - 0.5f)
            + dofScale * cam.up * (u01(rng) - 0.5f);
        
        segment.ray.origin = cam.position + posOffset;
        segment.ray.direction = glm::normalize(target - posOffset);


        assert(fabs(glm::length(segment.ray.direction)) - 1 < 0.01);


        segment.pixelIndex = index;
        segment.remainingBounces = traceDepth;
    }
}

// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
    int depth,
    int num_paths,
    PathSegment* pathSegments,
    Geom* geoms,
    int geoms_size,
    ShadeableIntersection* intersections,
    int* isValidIntersection,
    DeviceTexture* textures,
    int hdriIndex)
{
    int path_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (path_index < num_paths)
    {
        PathSegment& pathSegment = pathSegments[path_index];

        float t;
        glm::vec3 intersect_point;
        glm::vec3 normal;
        glm::vec2 uv;
        glm::vec3 tangent;
        glm::vec3 bitangent;
        float t_min = FLT_MAX;
        int hit_geom_index = -1;
        bool outside = true;

        glm::vec3 tmp_intersect;
        glm::vec3 tmp_normal;
        glm::vec2 tmp_uv;
        glm::vec3 tmp_tangent;
        glm::vec3 tmp_bitangent;

        // naive parse through global geoms

        for (int i = 0; i < geoms_size; i++)
        {
            Geom& geom = geoms[i];

            if (geom.type == CUBE)
            {
                t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
                assert(!isnan(tmp_normal.x));
                assert(!isnan(tmp_normal.y));
                assert(!isnan(tmp_normal.z));
            }
            else if (geom.type == SPHERE)
            {
                t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
                assert(!isnan(tmp_normal.x));
                assert(!isnan(tmp_normal.y));
                assert(!isnan(tmp_normal.z));
            }
            else if (geom.type == MESH) {

#if BVH
                t = meshIntersectionTestBVH(geom, pathSegment.ray, tmp_intersect, tmp_normal, tmp_uv, tmp_tangent, tmp_bitangent, outside);
#else
                t = meshIntersectionTestNaive(geom, pathSegment.ray, tmp_intersect, tmp_normal, tmp_uv, tmp_tangent, tmp_bitangent, outside);
#endif
            }
            // TODO: add more intersection tests here... triangle? metaball? CSG?

            // Compute the minimum t from the intersection tests to determine what
            // scene geometry object was hit first.
            if (t > 0.0f && t_min > t)
            {
                t_min = t;
                hit_geom_index = i;
                intersect_point = tmp_intersect;
                normal = tmp_normal;
                uv = tmp_uv;
                tangent = tmp_tangent;
                bitangent = tmp_bitangent;
            }
        }

        if (hit_geom_index == -1)
        {
            intersections[path_index].t = -1.0f;
            isValidIntersection[path_index] = 0;

            //HDRI eval;
            if (hdriIndex != -1) {
                pathSegment.ray.direction = glm::normalize(pathSegment.ray.direction);
                assert(fabs(glm::length(pathSegment.ray.direction) - 1.f) < 0.01f);

                float theta = atan2(pathSegment.ray.direction.x, pathSegment.ray.direction.z);
                float phi = asin(-pathSegment.ray.direction.y);

                float u = theta / (2 * PI) + 0.5;
                float v = phi / PI + 0.5;
                float4 hdriVal = tex2D<float4>(textures[hdriIndex].texHandle, u, v);
                pathSegment.radiance = glm::vec3(hdriVal.x, hdriVal.y, hdriVal.z);
            }

        }
        else
        {
            // The ray hits something
            intersections[path_index].t = t_min;
            intersections[path_index].materialId = geoms[hit_geom_index].materialid;
            assert(!isnan(normal.x));
            assert(!isnan(normal.y));
            assert(!isnan(normal.z));
            intersections[path_index].surfaceNormal = normal;
            intersections[path_index].uvCoord = uv;
            intersections[path_index].surfaceTangent = tangent;
            intersections[path_index].surfaceBitangent = bitangent;
            isValidIntersection[path_index] = 1;
        }
    }
}

// LOOK: "fake" shader demonstrating what you might do with the info in
// a ShadeableIntersection, as well as how to use thrust's random number
// generator. Observe that since the thrust random number generator basically
// adds "noise" to the iteration, the image should start off noisy and get
// cleaner as more iterations are computed.
//
// Note that this shader does NOT do a BSDF evaluation!
// Your shaders should handle that - this can allow techniques such as
// bump mapping.
__global__ void shadeFakeMaterial(
    int iter,
    int num_paths,
    ShadeableIntersection* shadeableIntersections,
    PathSegment* pathSegments,
    Material* materials,
    DeviceTexture* textures)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_paths)
    {
        pathSegments[idx].remainingBounces--;

        ShadeableIntersection intersection = shadeableIntersections[idx];
        if (intersection.t > 0.0f) // if the intersection exists...
        {
          // Set up the RNG
          // LOOK: this is how you use thrust's RNG! Please look at
          // makeSeededRandomEngine as well.
            thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, pathSegments[idx].remainingBounces);
            thrust::uniform_real_distribution<float> u01(0, 1);

            Material material = materials[intersection.materialId];
            glm::vec3 materialColor = material.baseColor;

            // If the material indicates that the object was a light, "light" the ray
            if (glm::length(material.emissive) > 0.0f) {
                pathSegments[idx].radiance = material.emissive;
                pathSegments[idx].remainingBounces = 0;
            }
            // Otherwise, do some pseudo-lighting computation. This is actually more
            // like what you would expect from shading in a rasterizer like OpenGL.
            else {
                //float lightTerm = glm::dot(intersection.surfaceNormal, glm::vec3(0.0f, 1.0f, 0.0f));
                //pathSegments[idx].color *= (materialColor * lightTerm) * 0.3f + ((1.0f - intersection.t * 0.02f) * materialColor) * 0.7f;
                //pathSegments[idx].color *= u01(rng); // apply some noise because why not
                assert(!isnan(intersection.surfaceNormal.x));
                assert(!isnan(intersection.surfaceNormal.y));
                assert(!isnan(intersection.surfaceNormal.z));
                scatterRay(pathSegments[idx], pathSegments[idx].ray.direction * intersection.t + pathSegments[idx].ray.origin,
                    intersection.surfaceNormal, material, textures, intersection.uvCoord, intersection.surfaceTangent,
                    intersection.surfaceBitangent, rng);
#if RUSSIANROULETTE
                // Russian Roulette
                float probDie = glm::clamp(glm::length(pathSegments[idx].throughput), 0.05f, 0.95f);
                if (u01(rng) > probDie) {
                    pathSegments[idx].remainingBounces = 0;
                }
                else {
                    pathSegments[idx].throughput /= probDie;
                }
#endif

            }
            // If there was no intersection, color the ray black.
            // Lots of renderers use 4 channel color, RGBA, where A = alpha, often
            // used for opacity, in which case they can indicate "no opacity".
            // This can be useful for post-processing and image compositing.
        }
        else {
            //pathSegments[idx].throughput = glm::vec3(1.0f);
            pathSegments[idx].remainingBounces = 0;

        }
    }
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3* image, glm::vec3* image_albedo, glm::vec3* image_normal, PathSegment* iterationPaths)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (index < nPaths)
    {
        PathSegment iterationPath = iterationPaths[index];
        image[iterationPath.pixelIndex] += iterationPath.throughput * iterationPath.radiance;
#if OIDN
        image_albedo[iterationPath.pixelIndex] = iterationPath.firstAlbedo;
        image_normal[iterationPath.pixelIndex] = iterationPath.firstNormal;
#endif 
    }
}


struct CompareIntersectionMaterials {
    template <typename Tuple>
    __host__ __device__ 
        bool operator()(const Tuple& a, const Tuple& b) const {
        // thrust::get<0>(a) retrieves the StructA from the tuple
        return thrust::get<0>(a).materialId < thrust::get<0>(b).materialId;
    }
};

struct GetKey {
    __host__ __device__
        int operator()(const ShadeableIntersection& s) const {
        return s.materialId;
    }
};

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4* pbo, int frame, int iter)
{
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    // 2D block for generating ray from camera
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
        (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
        (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // 1D block for path tracing
    const int blockSize1d = 128;

    ///////////////////////////////////////////////////////////////////////////

    // Recap:
    // * Initialize array of path rays (using rays that come out of the camera)
    //   * You can pass the Camera object to that kernel.
    //   * Each path ray must carry at minimum a (ray, color) pair,
    //   * where color starts as the multiplicative identity, white = (1, 1, 1).
    //   * This has already been done for you.
    // * For each depth:
    //   * Compute an intersection in the scene for each path ray.
    //     A very naive version of this has been implemented for you, but feel
    //     free to add more primitives and/or a better algorithm.
    //     Currently, intersection distance is recorded as a parametric distance,
    //     t, or a "distance along the ray." t = -1.0 indicates no intersection.
    //     * Color is attenuated (multiplied) by reflections off of any object
    //   * TODO: Stream compact away all of the terminated paths.
    //     You may use either your implementation or `thrust::remove_if` or its
    //     cousins.
    //     * Note that you can't really use a 2D kernel launch any more - switch
    //       to 1D.
    //   * TODO: Shade the rays that intersected something or didn't bottom out.
    //     That is, color the ray by performing a color computation according
    //     to the shader, then generate a new ray to continue the ray path.
    //     We recommend just updating the ray's PathSegment in place.
    //     Note that this step may come before or after stream compaction,
    //     since some shaders you write may also cause a path to terminate.
    // * Finally, add this iteration's results to the image. This has been done
    //   for you.

    // TODO: perform one iteration of path tracing

    generateRayFromCamera<<<blocksPerGrid2d, blockSize2d>>>(cam, iter, traceDepth, dev_paths);
    checkCUDAError("generate camera ray");

    int depth = 0;
    PathSegment* dev_path_end = dev_paths + pixelcount;
    int num_paths_total = dev_path_end - dev_paths;
    // --- PathSegment Tracing Stage ---
    // Shoot ray into scene, bounce between objects, push shading chunks

    bool iterationComplete = false;
    int num_paths = num_paths_total;
    while (!iterationComplete)
    {
        // clean shading chunks
        hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));
        hipMemset(dev_isValidIntersection, 0, pixelcount * sizeof(int));

        checkCUDAError("hipMemset");

        for (Geom geom : hst_scene->geoms) {
            if (geom.type == MESH) {
                assert(geom.mesh.onGpu);
            }
        }
        // tracing
        dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
        checkCUDAError("pre-intersection");

        computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
            depth,
            num_paths,
            dev_paths,
            dev_geoms,
            hst_scene->geoms.size(),
            dev_intersections,
            dev_isValidIntersection,
            dev_textures,
            hst_scene->hdriIndex
            );
        checkCUDAError("trace one bounce");
        hipDeviceSynchronize();
        depth++;

        //int* boolsHost = new int[num_paths];
        //hipMemcpy(boolsHost, dev_isValidIntersection, num_paths * sizeof(int), hipMemcpyDeviceToHost);
        //printf("Before: \n");
        //for (int i = 0; i < num_paths; ++i) {
        //    printf("%i , ", boolsHost[i]);
        //}
        //printf("\n");

        //PathSegment* paths = new PathSegment[num_paths];
        //hipMemcpy(paths, dev_paths, num_paths * sizeof(PathSegment), hipMemcpyDeviceToHost);
        //printf("Before: \n");
        //for (int i = 0; i < num_paths; ++i) {
        //    printf("%i , ", paths[i].pixelIndex);
        //}
        //printf("\n");


        //thrust::device_vector<PathSegment> v(dev_paths, dev_paths + num_paths);
        //thrust::device_vector<int>::iterator pivot = thurst::stable_partition(v.begin(), v.end(), function)
        //int count_hit = pivot - v.begin();
        //int count_miss = v.end() - pivot;

        //std::cout << "Even numbers: " << count_hit << std::endl;
        //std::cout << "Odd numbers: " << count_miss << std::endl;

        //// Print the result (copy to host to view)
        //thrust::host_vector<int> h_v = v;
        //for (PathSegment x : h_v) {
        //    std::cout << x.pixelIndex << " ";
        //}
        //std::cout << std::endl;

#if STREAMCOMPACTION
        num_paths = StreamCompaction::Efficient::partitionOnValidIntersect(num_paths, dev_paths, dev_isValidIntersection, dev_intersections);
        numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
        checkCUDAError("partition on intersect");
        if (num_paths == 0) break;
#endif


        //ShadeableIntersection* intersections = new ShadeableIntersection[num_paths];
        //hipMemcpy(intersections, dev_intersections, 100 * sizeof(ShadeableIntersection), hipMemcpyDeviceToHost);

        //for (int i = 0; i < 100; ++i) {
        //    if (intersections[i].uvCoord.x != 0.0) {
        //        printf("Intersection: %.2f, %.2f\n", intersections[i].uvCoord.x, intersections[i].uvCoord.y);
        //    }
        //}

        // TODO:
        // --- Shading Stage ---
        // Shade path segments based on intersections and generate new rays by
        // evaluating the BSDF.
        // Start off with just a big kernel that handles all the different
        // materials you have in the scenefile.
        // TODO: compare between directly shading the path segments and shading
        // path segments that have been reshuffled to be contiguous in memory.
#if MATERIALSORTING
        thrust::device_ptr<ShadeableIntersection> dev_thrust_intersections(dev_intersections);
        thrust::device_ptr<PathSegment> dev_thrust_paths(dev_paths);

        //auto zip_begin = thrust::make_zip_iterator(thrust::make_tuple(dev_thrust_intersections, dev_thrust_paths));
        //auto zip_end = thrust::make_zip_iterator(thrust::make_tuple(dev_thrust_intersections + num_paths, dev_thrust_paths + num_paths));
        
        thrust::device_vector<int> d_keys(num_paths);
        thrust::device_vector<int> d_keys2(num_paths);

        // thrust::transform extracts materialId from dev_intersections, and makes two copies
        thrust::transform(dev_thrust_intersections, dev_thrust_intersections + num_paths, d_keys.begin(), GetKey());
        thrust::transform(dev_thrust_intersections, dev_thrust_intersections + num_paths, d_keys2.begin(), GetKey());

        thrust::sort_by_key(d_keys.begin(), d_keys.end(), dev_thrust_paths);//zip_begin); ZIP IS SLOWER I THINK, TODO - check
        thrust::sort_by_key(d_keys2.begin(), d_keys2.end(), dev_thrust_intersections);

#endif
        shadeFakeMaterial<<<numblocksPathSegmentTracing, blockSize1d>>>(
            iter,
            num_paths,
            dev_intersections,
            dev_paths,
            dev_materials,
            dev_textures
        );
        checkCUDAError("shading");

#if STREAMCOMPACTION
        num_paths = StreamCompaction::Efficient::partitionOnBounces(num_paths, dev_paths);
        numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
        checkCUDAError("partition on bounces");

#endif


        //PathSegment* paths = new PathSegment[num_paths];
        //hipMemcpy(paths, dev_paths, num_paths * sizeof(PathSegment), hipMemcpyDeviceToHost);
        //printf("After: \n");
        //for (int i = 0; i < num_paths; ++i) {
        //    printf("%i, ", paths[i].remainingBounces);
        //}
        //printf("\n");
        //delete[] paths;

        if (depth >= traceDepth || num_paths <= 0)iterationComplete = true;
        //if (num_paths <= 0) iterationComplete = true; // TODO: should be based off stream compaction results.



        if (guiData != NULL)
        {
            guiData->TracedDepth = depth;
        }
    }

    // Assemble this iteration and apply it to the image
    dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
    finalGather<<<numBlocksPixels, blockSize1d>>>(pixelcount, dev_image, dev_image_albedo, dev_image_normal, dev_paths);

# if OIDN
    // denoise with OIDN
    oidn::DeviceRef device = oidn::newCUDADevice(0, NULL);
    device.commit();

    // Setup buffers
    oidn::BufferRef colorBuf = device.newBuffer(pixelcount * 3 * sizeof(float));
    oidn::BufferRef albedoBuf = device.newBuffer(pixelcount * 3 * sizeof(float));
    oidn::BufferRef normalBuf = device.newBuffer(pixelcount * 3 * sizeof(float));
    oidn::BufferRef outputBuf = device.newBuffer(pixelcount * 3 * sizeof(float));

    hipMemcpy(colorBuf.getData(), dev_image, pixelcount * 3 * sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(albedoBuf.getData(), dev_image_albedo, pixelcount * 3 * sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(normalBuf.getData(), dev_image_normal, pixelcount * 3 * sizeof(float), hipMemcpyDeviceToDevice);

    oidn::FilterRef filter = device.newFilter("RT"); // ray tracing filter
    filter.setImage("color", colorBuf, oidn::Format::Float3, cam.resolution.x, cam.resolution.y);
    filter.setImage("albedo", albedoBuf, oidn::Format::Float3, cam.resolution.x, cam.resolution.y);
    filter.setImage("normal", normalBuf, oidn::Format::Float3, cam.resolution.x, cam.resolution.y);
    filter.setImage("output", outputBuf, oidn::Format::Float3, cam.resolution.x, cam.resolution.y);
    filter.set("hdr", true);
    filter.commit();

    filter.execute();

    const char* errorMessage;
    if (device.getError(errorMessage) != oidn::Error::None) {
        printf("Error: ");
        printf(errorMessage);
    }

    hipMemcpy(dev_image_denoised, outputBuf.getData(), pixelcount * 3 * sizeof(float), hipMemcpyDeviceToDevice);
#endif

    ///////////////////////////////////////////////////////////////////////////

    // Send results to OpenGL buffer for rendering
#if OIDN
    sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image_denoised);
#else
    sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);
#endif
    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
        pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("pathtrace");
}
