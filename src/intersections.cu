#include "hip/hip_runtime.h"
#include "intersections.h"

__host__ __device__ float boxIntersectionTest(
    Geom box,
    Ray r,
    glm::vec3 &intersectionPoint,
    glm::vec3 &normal,
    bool &outside)
{
    Ray q;
    q.origin    =                multiplyMV(box.inverseTransform, glm::vec4(r.origin   , 1.0f));
    q.direction = glm::normalize(multiplyMV(box.inverseTransform, glm::vec4(r.direction, 0.0f)));

    float tmin = -1e38f;
    float tmax = 1e38f;
    glm::vec3 tmin_n;
    glm::vec3 tmax_n;
    for (int xyz = 0; xyz < 3; ++xyz)
    {
        float qdxyz = q.direction[xyz];
        /*if (glm::abs(qdxyz) > 0.00001f)*/
        {
            float t1 = (-0.5f - q.origin[xyz]) / qdxyz;
            float t2 = (+0.5f - q.origin[xyz]) / qdxyz;
            float ta = glm::min(t1, t2);
            float tb = glm::max(t1, t2);
            glm::vec3 n;
            n[xyz] = t2 < t1 ? +1 : -1;
            if (ta > 0 && ta > tmin)
            {
                tmin = ta;
                tmin_n = n;
            }
            if (tb < tmax)
            {
                tmax = tb;
                tmax_n = n;
            }
        }
    }

    if (tmax >= tmin && tmax > 0)
    {
        outside = true;
        if (tmin <= 0)
        {
            tmin = tmax;
            tmin_n = tmax_n;
            outside = false;
        }
        intersectionPoint = multiplyMV(box.transform, glm::vec4(getPointOnRay(q, tmin), 1.0f));
        normal = glm::normalize(multiplyMV(box.invTranspose, glm::vec4(tmin_n, 0.0f)));
        return glm::length(r.origin - intersectionPoint);
    }

    return -1;
}

__host__ __device__ float sphereIntersectionTest(
    Geom sphere,
    Ray r,
    glm::vec3 &intersectionPoint,
    glm::vec3 &normal,
    bool &outside)
{
    float radius = .5;

    glm::vec3 ro = multiplyMV(sphere.inverseTransform, glm::vec4(r.origin, 1.0f));
    glm::vec3 rd = glm::normalize(multiplyMV(sphere.inverseTransform, glm::vec4(r.direction, 0.0f)));

    Ray rt;
    rt.origin = ro;
    rt.direction = rd;

    float vDotDirection = glm::dot(rt.origin, rt.direction);
    float radicand = vDotDirection * vDotDirection - (glm::dot(rt.origin, rt.origin) - powf(radius, 2));
    if (radicand < 0)
    {
        return -1;
    }

    float squareRoot = sqrt(radicand);
    float firstTerm = -vDotDirection;
    float t1 = firstTerm + squareRoot;
    float t2 = firstTerm - squareRoot;

    float t = 0;
    if (t1 < 0 && t2 < 0)
    {
        return -1;
    }
    else if (t1 > 0 && t2 > 0)
    {
        t = min(t1, t2);
        outside = true;
    }
    else
    {
        t = max(t1, t2);
        outside = false;
    }

    glm::vec3 objspaceIntersection = getPointOnRay(rt, t);

    intersectionPoint = multiplyMV(sphere.transform, glm::vec4(objspaceIntersection, 1.f));
    normal = glm::normalize(multiplyMV(sphere.invTranspose, glm::vec4(objspaceIntersection, 0.f)));
    if (!outside)
    {
        normal = -normal;
    }

    return glm::length(r.origin - intersectionPoint);
}


__host__ __device__ float triangleIntersectionTest(
    glm::vec3 p1,
    glm::vec3 p2,
    glm::vec3 p3,
    Ray r,
    glm::vec3& intersectionPoint,
    glm::vec3& normal,
    bool& notBackface)
{

    assert(abs(glm::length(r.direction) - 1) < 0.01);
    glm::vec3 v12 = p2 - p1;
    glm::vec3 v13 = p3 - p1;
    normal = glm::normalize(glm::cross(v12, v13));

    // Find Ray intersection with plane of triangle
    float dist = glm::dot(p1 - r.origin, normal);
    // how much closer the ray move per time t?
    float step = glm::dot(r.direction, normal);
    if (abs(step) < 0.0001) {
        return -1; // Ray direction is parallel to the triangle
    }
    notBackface = true;
    if (step > 0) {
        normal *= -1; // I presume we want normal facing towards ray anyways
        notBackface = false;
    }

    float t = dist / step;

    if (t < 0) {
        return -1; // Plane intersection point is behind the ray origin - hmm. Or backface
    }

    intersectionPoint = r.origin + t * r.direction;

    // Barycentric check to determine if inside triangle.
    // Let planeIntersectionPoint be denoted as s.
    float areaS12 = abs(glm::length(glm::cross(v12, intersectionPoint - p1)));
    float areaS23 = abs(glm::length(glm::cross(p3 - p2, intersectionPoint - p2)));
    float areaS31 = abs(glm::length(glm::cross(intersectionPoint - p1, v13)));

    float area123 = abs(glm::length(glm::cross(v12, v13)));

    if (area123 - areaS12 - areaS23 - areaS31 < 0.0001) {
        return t;
    }
    return -1;

}

// TODO
__host__ __device__ float meshIntersectionTestNaive(
    Geom mesh,
    Ray r,
    glm::vec3& intersectionPoint,
    glm::vec3& normal,
    bool& outside)
{
    return -1;
}

// TODO
__host__ __device__ float meshIntersectionTestBVH(
    Geom mesh,
    Ray r,
    glm::vec3& intersectionPoint,
    glm::vec3& normal,
    bool& outside)
{
    return -1;
}